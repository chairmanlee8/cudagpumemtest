#include "gputests.h"

/************************************************************************************
 * Test 4 [Moving inversions, random pattern]
 * Test 4 uses the same algorithm as test 1 but the data pattern is a
 * random number and it's complement. This test is particularly effective
 * in finding difficult to detect data sensitive errors. A total of 60
 * patterns are used. The random number sequence is different with each pass
 * so multiple passes increase effectiveness.
 *
 *************************************************************************************/

int
test4(TestInputParams* tip, TestOutputParams* top, bool *term)
{
	unsigned int p1;
	//if (global_pattern == 0){
	p1 = get_random_num();
	//}else{
	//p1 = global_pattern;
	//}

	unsigned int p2 = ~p1;
	unsigned int err = 0;
	unsigned int iteration = 0;

	//DEBUG_PRINTF("Test4: Moving inversions test, with random pattern 0x%x and 0x%x\n", p1, p2);

repeat:
	err += move_inv_test(tip->ptr, tip->tot_num_blocks, p1, p2, top->err_vector, top->err_count, term);

	if (err == 0 && iteration == 0)
	{
		return hipSuccess;
	}
	if (iteration < tip->num_iterations)
	{
		//if(*term == true) break;
		//PRINTF("%dth repeating test4 because there are %d errors found in last run\n", iteration, err);
		iteration++;
		err = 0;
		if(*term == false) goto repeat;
	}

	return hipSuccess;
}