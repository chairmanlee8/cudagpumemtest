#include "hip/hip_runtime.h"
#include "gputests.h"

unsigned int get_random_num(void)
{
	/*struct timeval t0;
	if (gettimeofday(&t0, NULL) !=0){
	fprintf(stderr, "ERROR: gettimeofday() failed\n");
	exit(ERR_GENERAL);
	}*/

	//unsigned int seed= (unsigned int)t0.tv_sec;
	unsigned int seed = (unsigned int) GetTickCount();
	srand(seed);

	return rand();
}

unsigned long get_random_num_long(void)
{
	/*struct timeval t0;
	if (gettimeofday(&t0, NULL) !=0){
	fprintf(stderr, "ERROR: gettimeofday() failed\n");
	exit(ERR_GENERAL);
	}*/

	//unsigned int seed= (unsigned int)t0.tv_sec;
	unsigned int seed = (unsigned int) GetTickCount();
	srand(seed);

	unsigned int a = rand();
	unsigned int b = rand();

	unsigned long ret =  ((unsigned long)a) << 32;
	ret |= ((unsigned long)b);

	return ret;
}

/*__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
	unsigned int i;
	unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);
	if (ptr >= (unsigned int*) end_ptr)
	{
		return;
	}

	for (i = 0; i < BLOCKSIZE/sizeof(unsigned int); i++)
	{
		ptr[i] = pattern;
	}

	return;
}


__global__ void
kernel_move_inv_readwrite(char* _ptr, char* end_ptr, unsigned int p1, unsigned int p2, unsigned int* err,
                          unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
	unsigned int i;
	unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);
	if (ptr >= (unsigned int*) end_ptr)
	{
		return;
	}

	for (i = 0; i < BLOCKSIZE/sizeof(unsigned int); i++)
	{
		if (ptr[i] != p1)
		{
			RECORD_ERR(err, &ptr[i], p1, ptr[i]);
		}
		ptr[i] = p2;

	}

	return;
}


__global__ void
kernel_move_inv_read(char* _ptr, char* end_ptr,  unsigned int pattern, unsigned int* err,
                     unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read )
{
	unsigned int i;
	unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);
	if (ptr >= (unsigned int*) end_ptr)
	{
		return;
	}

	for (i = 0; i < BLOCKSIZE/sizeof(unsigned int); i++)
	{
		if (ptr[i] != pattern)
		{
			RECORD_ERR(err, &ptr[i], pattern, ptr[i]);
		}
	}

	return;
}


unsigned int
move_inv_test(char* ptr, unsigned int tot_num_blocks, unsigned int p1, unsigned p2, unsigned int* err_count, unsigned long* err_addr,
              unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read, bool *term)
{

	unsigned int i;
	unsigned int err = 0;
	char* end_ptr = ptr + tot_num_blocks* BLOCKSIZE;

	for (i= 0; i < tot_num_blocks; i+= GRIDSIZE)
	{
		if(*term == true) break;
		dim3 grid;
		grid.x= GRIDSIZE;
		kernel_move_inv_write<<<grid, 1>>>(ptr + i*BLOCKSIZE, end_ptr, p1); SYNC_CUERR;
		//SHOW_PROGRESS("move_inv_write", i, tot_num_blocks);
	}


	for (i=0; i < tot_num_blocks; i+= GRIDSIZE)
	{
		if(*term == true) break;
		dim3 grid;
		grid.x= GRIDSIZE;
		kernel_move_inv_readwrite<<<grid, 1>>>(ptr + i*BLOCKSIZE, end_ptr, p1, p2, err_count, err_addr, err_expect, err_current, err_second_read); SYNC_CUERR;
		//err += error_checking("move_inv_readwrite",  i);
		//SHOW_PROGRESS("move_inv_readwrite", i, tot_num_blocks);
	}

	for (i=0; i < tot_num_blocks; i+= GRIDSIZE)
	{
		if(*term == true) break;
		dim3 grid;
		grid.x= GRIDSIZE;
		kernel_move_inv_read<<<grid, 1>>>(ptr + i*BLOCKSIZE, end_ptr, p2, err_count, err_addr, err_expect, err_current, err_second_read); SYNC_CUERR;
		//err += error_checking("move_inv_read",  i);
		//SHOW_PROGRESS("move_inv_read", i, tot_num_blocks);
	}

	return err;

}*/