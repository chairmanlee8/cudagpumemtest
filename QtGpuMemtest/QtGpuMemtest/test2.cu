#include "gputests.h"

int
test2(char* ptr, unsigned int tot_num_blocks, int num_iterations, unsigned int* err, unsigned long* err_addr,
      unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read, bool *term)
{
	unsigned int p1 = 0;
	unsigned int p2 = ~p1;


	//DEBUG_PRINTF("Test2: Moving inversions test, with pattern 0x%x and 0x%x\n", p1, p2);
	move_inv_test(ptr, tot_num_blocks, p1, p2, err, err_addr, err_expect, err_current, err_second_read, term);
	//DEBUG_PRINTF("Test2: Moving inversions test, with pattern 0x%x and 0x%x\n", p2, p1);
	move_inv_test(ptr, tot_num_blocks, p2, p1, err, err_addr, err_expect, err_current, err_second_read, term);

	return hipSuccess;

}

